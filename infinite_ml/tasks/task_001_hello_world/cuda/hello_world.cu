#include "hip/hip_runtime.h"
#include <stdio.h>
#include <hip/hip_runtime.h>
#include "hello_world.cuh"

// CUDA kernel that prints a Hello World message
__global__ void helloWorldKernel() {
    printf("Hello World from thread %d in block %d!\n", threadIdx.x, blockIdx.x);
}

void helloWorldCuda() {
    // Print message from host (CPU)
    printf("Hello World from the host (CPU)!\n");
    
    // Launch kernel with 2 blocks and 4 threads per block
    helloWorldKernel<<<2, 4>>>();
    
    // Synchronize to ensure all printf statements from the kernel are displayed
    hipDeviceSynchronize();
    
    // Check for errors
    hipError_t error = hipGetLastError();
    if (error != hipSuccess) {
        printf("CUDA Error: %s\n", hipGetErrorString(error));
        return;
    }
    
    printf("CUDA kernel execution completed successfully!\n");
    return;
}


int main() {
    // Print message from host (CPU)
    printf("Hello World from the host (CPU)!\n");
    
    // Launch kernel with 2 blocks and 4 threads per block
    helloWorldKernel<<<2, 4>>>();
    
    // Synchronize to ensure all printf statements from the kernel are displayed
    hipDeviceSynchronize();
    
    // Check for errors
    hipError_t error = hipGetLastError();
    if (error != hipSuccess) {
        printf("CUDA Error: %s\n", hipGetErrorString(error));
        return -1;
    }
    
    printf("CUDA kernel execution completed successfully!\n");
    return 0;
}