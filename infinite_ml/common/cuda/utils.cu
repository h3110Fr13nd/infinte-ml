#include "utils.cuh"
#include <hip/hip_runtime.h>

// Simple utility to check if CUDA is available
bool isCudaAvailable() {
    int deviceCount = 0;
    hipError_t error = hipGetDeviceCount(&deviceCount);
    return (error == hipSuccess && deviceCount > 0);
}

// Get total device memory in bytes - remove default argument here
size_t getDeviceMemory(int deviceId) {
    hipDeviceProp_t prop;
    if (hipGetDeviceProperties(&prop, deviceId) != hipSuccess) {
        return 0;
    }
    return prop.totalGlobalMem;
}

// Synchronize device (wrapper for hipDeviceSynchronize)
void syncDevice() {
    hipDeviceSynchronize();
}
